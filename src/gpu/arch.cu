#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>

#include "../global.hpp"
#include "../getarg.hpp"

#include "peakperf_helper_cuda.hpp"
#include "kernel.hpp"

enum {
  ARCH_FERMI,
  ARCH_KEPLER,
  ARCH_MAXWELL,
  ARCH_PASCAL,
  ARCH_VOLTA,
  ARCH_TURING,
  ARCH_AMPERE,
  ARCH_ADA,
  ARCH_UNKNOWN
};

static const char *uarch_str[] = {
  /*[ARCH_FERMI]      = */ "Fermi",
  /*[ARCH_KEPLER]     = */ "Kepler",
  /*[ARCH_MAXWELL]    = */ "Maxwell",
  /*[ARCH_PASCAL]     = */ "Pascal",
  /*[ARCH_VOLTA]      = */ "Volta",
  /*[ARCH_TURING]     = */ "Turing",
  /*[ARCH_AMPERE]     = */ "Ampere",
  /*[ARCH_ADA]        = */ "Ada",
};

struct benchmark_gpu {
  int nbk; // Blocks per thread
  int tpb; // Threads per block
  int n;
  double gflops;
  const char* name;
  float *d_A;
  float *d_B;
  float *d_C;
};

// We assume only one gpu is present...
struct gpu {
  int compute_capability;
  int latency;
  int sm_count;
  int cc_major;
  int cc_minor;
  char uarch;
  char* name;
};

void print_cuda_gpus_list() {
  hipError_t err = hipSuccess;
  int num_gpus = -1;
  if ((err = hipGetDeviceCount(&num_gpus)) != hipSuccess) {
    printErr("%s: %s", hipGetErrorName(err), hipGetErrorString(err));
    return;
  }
  printf("GPUs available: %d\n", num_gpus);

  if(num_gpus > 0) {
    hipDeviceProp_t deviceProp;
    int max_len = 0;

    for(int idx=0; idx < num_gpus; idx++) {
      hipGetDeviceProperties(&deviceProp, idx);
      max_len = max(max_len, (int) strlen(deviceProp.name));
    }

    for(int i=0; i < max_len + 28; i++) putchar('-');
    putchar('\n');
    for(int idx=0; idx < num_gpus; idx++) {
      hipGetDeviceProperties(&deviceProp, idx);
      printf("%d: %s (Compute Capability %d.%d)\n", idx, deviceProp.name, deviceProp.major, deviceProp.minor);
    }
  }
}

struct gpu* get_gpu_info(int gpu_idx) {
  hipError_t err = hipSuccess;
  struct gpu* gpu = (struct gpu *) malloc(sizeof(struct gpu));

  int num_gpus = -1;
  if ((err = hipGetDeviceCount(&num_gpus)) != hipSuccess) {
    printErr("%s: %s", hipGetErrorName(err), hipGetErrorString(err));
    return NULL;
  }
  if(num_gpus <= 0) {
    printErr("No CUDA capable devices found!");
    return NULL;
  }
  if(gpu_idx < 0) {
    printErr("GPU index must be equal or greater than zero");
    return NULL;
  }
  if(gpu_idx+1 > num_gpus) {
    printErr("Requested GPU index %d in a system with %d GPUs", gpu_idx, num_gpus);
    return NULL;
  }

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, gpu_idx);

  int gpu_name_len = strlen(deviceProp.name);
  gpu->cc_major = deviceProp.major;
  gpu->cc_minor = deviceProp.minor;
  gpu->compute_capability = deviceProp.major * 10 + deviceProp.minor;
  gpu->sm_count = deviceProp.multiProcessorCount;
  gpu->name = (char *) malloc(sizeof(char) * (gpu_name_len + 1));
  memset(gpu->name, 0, gpu_name_len + 1);
  strcpy(gpu->name, deviceProp.name);

  // https://en.wikipedia.org/w/index.php?title=CUDA#GPUs_supported
  switch(gpu->compute_capability) {
    case 20:
    case 21:
      gpu->uarch = ARCH_FERMI;
      break;
    case 30:
    case 32:
    case 35:
    case 37:
      gpu->uarch = ARCH_KEPLER;
      break;
    case 50:
    case 52:
    case 53:
      gpu->uarch = ARCH_MAXWELL;
      break;
    case 60:
    case 61:
    case 62:
      gpu->uarch = ARCH_PASCAL;
      break;
    case 70:
    case 72:
      gpu->uarch = ARCH_VOLTA;
      break;
    case 75:
      gpu->uarch = ARCH_TURING;
      break;
    case 80:
    case 86:
    case 87:
      gpu->uarch = ARCH_AMPERE;
      break;
    case 89:
      gpu->uarch = ARCH_ADA;
      break;
    default:
      printf("GPU: %s\n", gpu->name);
      printErr("Invalid uarch: %d.%d\n", deviceProp.major, deviceProp.minor);
      return NULL;
  }

  // https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#arithmetic-instructions (?)
  switch(gpu->uarch) {
    case ARCH_FERMI:      // UNTESTED
    case ARCH_KEPLER:     // UNTESTED
    case ARCH_MAXWELL:
    case ARCH_PASCAL:
    case ARCH_VOLTA:      // UNTESTED
      gpu->latency = 6;
      break;
    case ARCH_TURING:
    case ARCH_AMPERE:     // UNTESTED
    case ARCH_ADA:     // UNTESTED
      gpu->latency = 4;
      break;
    default:
      printErr("latency unknown for uarch: %d.%d\n", deviceProp.major, deviceProp.minor);
      return NULL;
  }

  return gpu;
}

struct benchmark_gpu* init_benchmark_gpu(struct gpu* gpu, int nbk, int tpb) {
  struct benchmark_gpu* bench = (struct benchmark_gpu *) malloc(sizeof(struct benchmark_gpu));

  // TODO: Warn if nbk or tpb are not optimal values
  if(gpu->compute_capability >= 50) {
    bench->nbk = (nbk == INVALID_CFG) ? gpu->sm_count : nbk;
    bench->tpb = (tpb == INVALID_CFG) ? (gpu->latency * _ConvertSMVer2Cores(gpu->cc_major, gpu->cc_minor)): tpb;
  }
  else {
    // Fix for old architectures where too many tpb were launched (this config is supposed to keep the same performance)
    bench->nbk = (nbk == INVALID_CFG) ? (gpu->latency * gpu->sm_count) : nbk;
    bench->tpb = (tpb == INVALID_CFG) ? _ConvertSMVer2Cores(gpu->cc_major, gpu->cc_minor) : tpb;
  }
  bench->n = 16 * bench->nbk * bench->tpb;
  bench->gflops = (double)(BENCHMARK_GPU_ITERS * 2 * (long)bench->n)/(long)1000000000;

  hipError_t err = hipSuccess;
  float *h_A;
  float *h_B;
  int size = bench->n * sizeof(float);

  hipSetDevice(0);

  if ((err = hipHostMalloc((void **)&h_A, size)) != hipSuccess) {
    printErr("XXX %s: %s", hipGetErrorName(err), hipGetErrorString(err));
    return NULL;
  }

  if ((err = hipHostMalloc((void **)&h_B, size)) != hipSuccess) {
    printErr("XXX %s: %s", hipGetErrorName(err), hipGetErrorString(err));
    return NULL;
  }

  for (int i = 0; i < bench->n; i++) {
    h_A[i] = rand()/(float)RAND_MAX;
    h_B[i] = rand()/(float)RAND_MAX;
  }


  if ((err = hipMalloc((void **) &(bench->d_A), size)) != hipSuccess) {
    printErr("%s: %s", hipGetErrorName(err), hipGetErrorString(err));
    return NULL;
  }

  if ((err = hipMalloc((void **) &(bench->d_B), size)) != hipSuccess) {
    printErr("%s: %s", hipGetErrorName(err), hipGetErrorString(err));
    return NULL;
  }

  if ((err = hipMalloc((void **) &(bench->d_C), size)) != hipSuccess) {
    printErr("%s: %s", hipGetErrorName(err), hipGetErrorString(err));
    return NULL;
  }

  if ((err = hipMemcpy(bench->d_A, h_A, size, hipMemcpyHostToDevice)) != hipSuccess) {
    printErr("%s: %s", hipGetErrorName(err), hipGetErrorString(err));
    return NULL;
  }

  if ((err = hipMemcpy(bench->d_B, h_B, size, hipMemcpyHostToDevice)) != hipSuccess) {
    printErr("%s: %s", hipGetErrorName(err), hipGetErrorString(err));
    return NULL;
  }

  return bench;
}

double get_gflops_gpu(struct benchmark_gpu* bench) {
  return bench->gflops;
}

bool compute_gpu(struct benchmark_gpu* bench, double* e_time) {
  hipError_t err = hipSuccess;
  hipEvent_t start;
  hipEvent_t stop;
  dim3 dimGrid(bench->nbk, 1, 1);
  dim3 dimBlock(bench->tpb, 1, 1);

  hipDeviceSynchronize();

  if ((err = hipEventCreate(&start)) != hipSuccess) {
    printErr("%s: %s", hipGetErrorName(err), hipGetErrorString(err));
    return false;
  }
  if ((err = hipEventCreate(&stop)) != hipSuccess) {
    printErr("%s: %s", hipGetErrorName(err), hipGetErrorString(err));
    return false;
  }

  hipEventRecord(start, 0);
  compute_kernel<<<dimGrid, dimBlock>>>(bench->d_A, bench->d_B, bench->d_C, bench->n);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  float e_time_gpu;
  hipEventElapsedTime(&e_time_gpu, start, stop);
  *e_time = e_time_gpu/1000;

  if ((err = hipGetLastError()) != hipSuccess) {
    printErr("%s: %s", hipGetErrorName(err), hipGetErrorString(err));
    if(err == hipErrorLaunchTimeOut) {
      printf("         NOTE: The GPU used by peakperf is attached to a display.\n");
      printf("         A possible workaround is to stop X server by issuing:\n");
      printf("         sudo systemctl isolate multi-user.target\n");
    }
    return false;
  }
  return true;
}

void exit_benchmark_gpu() {
  hipDeviceReset();
}

char* get_str_gpu_name(struct gpu* gpu) {
  return gpu->name;
}

const char* get_str_gpu_uarch(struct gpu* gpu) {
  return uarch_str[gpu->uarch];
}

int get_n_blocks(struct benchmark_gpu* bench) {
  return bench->nbk;
}

int get_threads_per_block(struct benchmark_gpu* bench) {
  return bench->tpb;
}
