#include "hip/hip_runtime.h"
#include "kernel.hpp"
#include <stdio.h>
#include <stdint.h>
#define N 16
#define gid threadIdx.x + blockIdx.x * blockDim.x
#define off gid*N


__global__
void compute_kernel(float *vec_a, float *vec_b, float *vec_c, int n) {
  __shared__ float myblockA[N];
  __shared__ float myblockB[N];
  __shared__ float myblockC[N];

  #pragma unroll
  for(int i = 0; i < N; i++){
   myblockA[i] = vec_a[off+i];
   myblockB[i] = vec_b[off+i];
   myblockC[i] = vec_a[off+i];
  }

  __syncthreads();

  #pragma unroll 32
  for(long i=0; i < BENCHMARK_GPU_ITERS; i++) {
    #pragma unroll
    for(int j = 0; j < N; j++){
      myblockC[j] = (myblockC[j] * myblockA[j]) + myblockB[j];
    }
  }

  #pragma unroll
  for(int i = 0; i < N; i++){
    vec_c[off+i] = myblockC[i];
  }

}

